#include "hip/hip_runtime.h"
/****************************************************************************
* Nome:                  Davide
* Cognome:               Conti
* Numero di matricola:   0000806467
****************************************************************************/

/****************************************************************************
 *
 * earthquake.c - Simple 2D earthquake model
 *
 * Copyright (C) 2018 Moreno Marzolla <moreno.marzolla(at)unibo.it>
 * Last updated on 2018-12-29
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * ---------------------------------------------------------------------------
 *
 * Versione di riferimento del progetto di High Performance Computing
 * 2018/2019, corso di laurea in Ingegneria e Scienze Informatiche,
 * Universita' di Bologna. Per una descrizione del modello si vedano
 * le specifiche sulla pagina del corso:
 *
 * http://moreno.marzolla.name/teaching/HPC/
 *
 * Per compilare:
 *
 * gcc -D_XOPEN_SOURCE=600 -std=c99 -Wall -Wpedantic earthquake.c -o earthquake
 *
 * (il flag -D_XOPEN_SOURCE=600 e' superfluo perche' viene settato
 * nell'header "hpc.h", ma definirlo tramite la riga di comando fa si'
 * che il programma compili correttamente anche se inavvertitamente
 * non si include "hpc.h", o per errore non lo si include come primo
 * file come necessario).
 *
 * Per eseguire il programma si puo' usare la riga di comando seguente:
 *
 * ./earthquake 100000 256 > out
 *
 * Il primo parametro indica il numero di timestep, e il secondo la
 * dimensione (lato) del dominio. L'output consiste in coppie di
 * valori numerici (100000 in questo caso) il cui significato e'
 * spiegato nella specifica del progetto.
 *
 ****************************************************************************/
#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>     /* rand() */
#include <assert.h>

/* energia massima */
#define EMAX 4.0f
/* energia da aggiungere ad ogni timestep */
#define EDELTA 1e-4
#define BLKDIM 32
#define BLKSIZE 1024

/**
 * Restituisce un puntatore all'elemento di coordinate (i,j) del
 * dominio grid con n colonne.
 */
__device__ __host__ static inline float *IDX(float *grid, int i, int j, int n)
{
    return (grid + i*n + j);
}

/**
 * Restituisce un numero reale pseudocasuale con probabilita' uniforme
 * nell'intervallo [a, b], con a < b.
 */
float randab( float a, float b )
{
    return a + (b-a)*(rand() / (float)RAND_MAX);
}

/**
 * Inizializza il dominio grid di dimensioni n*n con valori di energia
 * scelti con probabilità uniforme nell'intervallo [fmin, fmax], con
 * fmin < fmax.
 *
 * NON PARALLELIZZARE QUESTA FUNZIONE: rand() non e' thread-safe,
 * qundi non va usata in blocchi paralleli OpenMP; inoltre la funzione
 * non si "comporta bene" con MPI (i dettagli non sono importanti, ma
 * posso spiegarli a chi e' interessato). Di conseguenza, questa
 * funzione va eseguita dalla CPU, e solo dal master (se si usa MPI).
 */
void setup( float* grid, int n, float fmin, float fmax )
{
    for (int i = 0; i < n; i++ ) {
        for (int j = 0; j < n; j++ ) {
            *IDX(grid, i, j, n) = randab(fmin, fmax);
        }
    }
}

/**
 * Somma delta a tutte le celle del dominio grid di dimensioni
 * n*n. Questa funzione realizza il passo 1 descritto nella specifica
 * del progetto.
 */
__global__ void increment_energy( float *grid, int n, float delta )
{

    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && j < n) {
        *IDX(grid, i, j, n) += delta;
    }
}

/**
 * Restituisce il numero di celle la cui energia e' strettamente
 * maggiore di EMAX.
 */
__global__ void count_cells(float *grid, int n, int *c)
{
    __shared__ int partial_sums[BLKSIZE];
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    const int lindex = blockDim.x * threadIdx.y + threadIdx.x;
    const int gindex = i * n + j;
    const int numberOfElements = n * n;
    int bsize = (blockDim.x * blockDim.y) / 2;
    if (gindex < numberOfElements) {
        partial_sums[lindex] = (*IDX(grid, i, j, n)) > EMAX ? 1 : 0;
    } else {
        partial_sums[lindex] = 0;
    }
    __syncthreads();
    while (bsize > 0) {
        if (lindex < bsize) {
            partial_sums[lindex] += partial_sums[lindex + bsize];
        }
        bsize = bsize / 2;
        __syncthreads();
    }
    if (lindex == 0) {
        atomicAdd(c, partial_sums[0]);
    }
}

/**
 * Distribuisce l'energia di ogni cella a quelle adiacenti (se
 * presenti). cur denota il dominio corrente, next denota il dominio
 * che conterra' il nuovo valore delle energie. Questa funzione
 * realizza il passo 2 descritto nella specifica del progetto.
 */
__global__ void propagate_energy(float *cur, float *next, int n)
 {
     const float FDELTA = EMAX/4;
     const int i = blockIdx.y * blockDim.y + threadIdx.y;
     const int j = blockIdx.x * blockDim.x + threadIdx.x;

     if (i < n && j < n) {
         float F = *IDX(cur, i, j, n);

         /* Se l'energia del vicino di sinistra (se esiste) e'
            maggiore di EMAX, allora la cella (i,j) ricevera'
            energia addizionale FDELTA = EMAX/4 */
         if ((j > 0) && (*IDX(cur, i, j - 1, n) > EMAX)) { F += FDELTA; }
         /* Idem per il vicino di destra */
         if ((j<n-1) && (*IDX(cur, i, j + 1, n) > EMAX)) { F += FDELTA; }
         /* Idem per il vicino in alto */
         if ((i>0) && (*IDX(cur, i - 1, j, n) > EMAX)) { F += FDELTA; }
         /* Idem per il vicino in basso */
         if ((i<n-1) && (*IDX(cur, i + 1, j, n) > EMAX)) { F += FDELTA; }

         /* Si noti che il valore di F potrebbe essere ancora
            maggiore di EMAX; questo non e' un problema:
            l'eventuale eccesso verra' rilasciato al termine delle
            successive iterazioni fino a riportare il valore
            dell'energia sotto la foglia EMAX. */
         if (F > EMAX) {
             F -= EMAX;
         }

         *IDX(next, i, j, n) = F;
     }
}

/**
 * Restituisce l'energia media delle celle del dominio grid di
 * dimensioni n*n. Il dominio non viene modificato.
 */
__global__ void average_energy(float *grid, int n, float *sum)
 {
     __shared__ float partial_sums[BLKSIZE];
     const int i = blockIdx.y * blockDim.y + threadIdx.y;
     const int j = blockIdx.x * blockDim.x + threadIdx.x;
     const int lindex = blockDim.x * threadIdx.y + threadIdx.x;
     const int gindex = i * n + j;
     const int numberOfElements = n * n;
     int bsize = (blockDim.x * blockDim.y) / 2;
     if (gindex < numberOfElements) {
         partial_sums[lindex] = *IDX(grid, i, j, n);
     } else {
         partial_sums[lindex] = 0;
     }
     __syncthreads();
     while (bsize > 0) {
         if (lindex < bsize) {
             partial_sums[lindex] += partial_sums[lindex + bsize];
         }
         bsize = bsize / 2;
         __syncthreads();
     }
     if (lindex == 0) {
         atomicAdd(sum, partial_sums[0]);
     }
}

int main( int argc, char* argv[] )
{
    const int zero = 0;
    const float zero1 = 0.0f;

    float *cur, *next;
    float *d_cur, *d_next, *d_sum;
    int *d_c;
    int s, n = 256, nsteps = 2048;
    float Emean;
    int c;

    srand(19); /* Inizializzazione del generatore pseudocasuale */

    if ( argc > 3 ) {
        fprintf(stderr, "Usage: %s [nsteps [n]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        nsteps = atoi(argv[1]);
    }

    if ( argc > 2 ) {
        n = atoi(argv[2]);
    }

    const size_t size = n * n * sizeof(float);
    /* Allochiamo i domini */
    cur = (float*)malloc(size); assert(cur);
    next = (float*)malloc(size); assert(next);

    /* L'energia iniziale di ciascuna cella e' scelta
       con probabilita' uniforme nell'intervallo [0, EMAX*0.1] */
    setup(cur, n, 0, EMAX*0.1);

    dim3 block(BLKDIM, BLKDIM);
    dim3 grid((n + BLKDIM - 1) / BLKDIM, ( n + BLKDIM - 1) / BLKDIM);

    hipMalloc((void**)&d_cur, size);
    hipMalloc((void**)&d_next, size);
    hipMalloc((void**)&d_c, sizeof(*d_c));
    hipMalloc((void**)&d_sum, sizeof(*d_sum));

    hipMemcpy(d_cur, cur, size, hipMemcpyHostToDevice);
    hipMemcpy(d_next, next, size, hipMemcpyHostToDevice);

    const double tstart = hpc_gettime();
    for (s=0; s<nsteps; s++) {
        /* L'ordine delle istruzioni che seguono e' importante */
        increment_energy<<<grid, block>>>(d_cur, n, EDELTA);
        hipMemcpy(d_c, &zero, sizeof(zero), hipMemcpyHostToDevice);
        count_cells<<<grid, block>>>(d_cur, n, d_c);
        hipMemcpy(&c, d_c, sizeof(c), hipMemcpyDeviceToHost);
        propagate_energy<<<grid, block>>>(d_cur, d_next, n);
        hipMemcpy(d_sum, &zero1, sizeof(zero1), hipMemcpyHostToDevice);
        average_energy<<<grid, block>>>(d_next, n, d_sum);
        hipMemcpy(&Emean, d_sum, sizeof(Emean), hipMemcpyDeviceToHost);
        Emean /= (n*n);

        printf("%d %f\n", c, Emean);

        float *tmp = d_cur;
        d_cur = d_next;
        d_next = tmp;
    }
    const double elapsed = hpc_gettime() - tstart;

    double Mupdates = (((double)n)*n/1.0e6)*nsteps; /* milioni di celle aggiornate per ogni secondo di wall clock time */
    fprintf(stderr, "%s : %.4f Mupdates in %.4f seconds (%f Mupd/sec)\n", argv[0], Mupdates, elapsed, Mupdates/elapsed);

    /* Libero la memoria */
    free(cur);
    free(next);
    hipFree(d_cur);
    hipFree(d_next);
    hipFree(d_c);
    hipFree(d_sum);
    return EXIT_SUCCESS;
}
